#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int resX,int  maxIterations, int* device_img, size_t pitch) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int x_dim = blockIdx.x * blockDim.x + threadIdx.x;
    int y_dim = blockIdx.y * blockDim.y + threadIdx.y;

    float c_re = lowerX + x_dim * stepX;
    float c_im = lowerY + y_dim * stepY;

    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < maxIterations; ++i) {

        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }
    
    *((int *)((void *)device_img + y_dim * pitch) + x_dim) = i;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int *host_img;
    int *device_img;
    size_t pitch;
    hipHostAlloc(&host_img, resX * resY * sizeof(int), hipHostMallocDefault);
    hipMallocPitch(&device_img, &pitch, resX * sizeof(int), resY);

    dim3 block(16, 16);
    dim3 grid(resX / block.x, resY / block.y);
    mandelKernel<<<grid, block>>>(lowerX, lowerY, stepX, stepY, resX, maxIterations, device_img, pitch);

    hipMemcpy2D(host_img, resX * sizeof(int), device_img, pitch, resX * sizeof(int), resY, hipMemcpyDefault);
    hipFree(device_img);
    memcpy(img, host_img, resX * resY * sizeof(int));
    hipHostFree(host_img);
}
